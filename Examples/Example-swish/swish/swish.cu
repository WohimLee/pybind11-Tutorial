#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

// cu 会交给 nvcc 编译
// cpp 会交给 gcc 编译

static __global__ void swish_forward(float* x, float* y, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= n) return;
    // swish
    y[i] = x[i] * (1 + exp(-x[i]));
}

at::Tensor forward(const at::Tensor x){
    // 设置当前的 device id
    const at::cuda::OptionalCUDAGuard device_guard(device_of(x));
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    // auto options = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    at::Tensor y = torch::zeros_like(x);

    dim3 block(512);
    dim3 grid((x.numel() + block.x - 1) / block.x);
    swish_forward<<<grid, block, 0, stream>>>(
        x.data_ptr<float>(), y.data_ptr<float>(), x.numel()
    );

    printf("x.shape = %d x %d\n", x.size(0), x.size(1));
    return y;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m){
    m.def("forward", &forward); // 定义函数名
}

